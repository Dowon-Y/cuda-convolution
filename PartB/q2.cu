
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__ void add(int n, float *x, float *y){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
	y[i] = x[i] + y[i];
}

int main(int argc, char** argv) {
	int K = 1;
	int N = 1<<20;
	int grid_size = 1;
	int block_size = 1;

    if (argc == 4) {
        sscanf(argv[1], "%d", &K);
		sscanf(argv[2], "%d", &grid_size);
		sscanf(argv[3], "%d", &block_size);
    }
	std::cout << "-------------------------------------" << std::endl;
	std::cout << "K: " << K << ", grid size: " << grid_size << ", block size: " << block_size << std::endl;
	N = K * N;
	size_t size = N * sizeof(float);
	float *x, *y;
	// Allocate input vectors h_A and h_B in host memory
	float* hx = (float*)malloc(size);
	float* hy = (float*)malloc(size);

	// initialize x and y arrays on the host
	for (int i = 0; i < N; i++) {
		hx[i] = 1.0f;
		hy[i] = 2.0f;
	}

	// Allocate vectors in device memory
	hipMalloc(&x, size);
	hipMalloc(&y, size);
	// Copy vectors from host memory to device global memory
	hipMemcpy(x, hx, size, hipMemcpyHostToDevice);
	hipMemcpy(y, hy, size, hipMemcpyHostToDevice);

	
	dim3 dimGrid(grid_size);
	dim3 dimBlock(block_size);

	// invoke kernel
	add<<<dimGrid, dimBlock>>>(N, x, y);
	hipMemcpy(hy, y, size, hipMemcpyDeviceToHost);

	// Check for errors (all values should be 3.0f)
	float maxError = 0.0f;
	for (int i = 0; i < N; i++) {
		maxError = fmax(maxError, fabs(hy[i]-3.0f));
	}
	std::cout << "Max error: " << maxError << std::endl;
	// Free memory
	hipFree(x); hipFree(y);
	free(hx); free(hy);
	return 0;
}
